#include "hip/hip_runtime.h"
#include "contrast.cuh"
#include "utils.cuh"
#include <cmath>

#define BATCH_DIM 32
#define STREAM_COUNT BATCH_DIM

__global__ static void computeHistogram_kernel(int *histogram, const uchar *inputPixels, int size) {
    int start = (blockDim.x * blockIdx.x + threadIdx.x) * BATCH_DIM;
    int stop = start + BATCH_DIM;

    for (int i = start; (i < stop) && (i < size); ++i) {
        atomicAdd(&histogram[inputPixels[i]], 1);
    }
}

static hipError_t computeHistogram(int **dev_outputHistogram, int histogramRange, uchar **dev_inputPixels, uchar *inputPixels, int rows, int cols) {
    hipError_t cudaStatus;
    hipStream_t streams[STREAM_COUNT];
    int blocks = ceil((double) rows / STREAM_COUNT);
    int threads = ceil((double) cols / BATCH_DIM);
    int size = 0;

    cudaStatus = cudaCalloc(dev_outputHistogram, histogramRange * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "outputHistogram cudaCalloc failed!\n");
        return cudaStatus;
    }

    cudaStatus = hipMalloc(dev_inputPixels, rows * cols * sizeof(uchar));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_inputPixels hipMalloc failed!\n");
        return cudaStatus;
    }

    for (int i = 0; i < STREAM_COUNT; ++i) {
        hipStreamCreate(&streams[i]);

        if (i * blocks < rows) {
            size = (((i + 1) * blocks * cols) < (rows * cols) ? (blocks * cols) : ((rows - i * blocks) * cols));

            cudaStatus = hipMemcpyAsync(&dev_inputPixels[0][i * blocks * cols],
                                         &inputPixels[i * blocks * cols],
                                         size * sizeof(uchar),
                                         hipMemcpyHostToDevice,
                                         streams[i]);
            if (cudaStatus != hipSuccess) {
                fprintf(stderr, "dev_inputPixels[%d] hipMemcpyAsync failed!\n", i);
                return cudaStatus;
            }

            computeHistogram_kernel <<<blocks, threads, 0, streams[i]>>>(dev_outputHistogram[0], &dev_inputPixels[0][i * blocks * cols], size);
        }
    }

    // hipDeviceSynchronize waits for the kernel to finish
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching computeHistogram_kernel!\n", cudaStatus);
    }

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Failed to launch computeHistogram_kernel: %s\n", hipGetErrorString(cudaStatus));
    }

    for (int i = 0; i < STREAM_COUNT; ++i) {
        hipStreamDestroy(streams[i]);
    }

    return cudaStatus;
}

__global__ static void computeCumulativeSum_kernel(int *cumulativeHistogram, const int *histogram) {
    extern __shared__ int sharedHistogram[];
    unsigned int id = threadIdx.x;

    sharedHistogram[id] = histogram[id];

    __syncthreads();

    for (int i = 0; i <= id; ++i) {
        cumulativeHistogram[id] += sharedHistogram[i];
    }
}

static hipError_t computeCumulativeSum(int **dev_outputCumulativeHistogram, int *dev_histogram, int histogramRange) {
    hipError_t cudaStatus;

    cudaStatus = cudaCalloc(dev_outputCumulativeHistogram, histogramRange * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_outputCumulativeHistogram cudaCalloc failed!\n");
        return cudaStatus;
    }

    computeCumulativeSum_kernel<<<1, histogramRange, histogramRange * sizeof(int)>>>(dev_outputCumulativeHistogram[0], dev_histogram);

    // hipDeviceSynchronize waits for the kernel to finish
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching computeCumulativeSum_kernel!\n", cudaStatus);
    }

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Failed to launch computeHistogram_kernel: %s\n", hipGetErrorString(cudaStatus));
    }

    return cudaStatus;
}

__global__ static void equalizeHistogram_kernel(int *pixelMapping, const int *cumulativeHistogram, int histogramRange) {
    extern __shared__ int sharedCumulativeHistogram[];
    unsigned int cumulativeHistogramSize = histogramRange;
    unsigned int id = threadIdx.x;

    sharedCumulativeHistogram[id] = cumulativeHistogram[id];

    __syncthreads();

    pixelMapping[id] = (int) (sharedCumulativeHistogram[id] * (cumulativeHistogramSize - 1) /
                              sharedCumulativeHistogram[cumulativeHistogramSize - 1]);
}


static hipError_t equalizeHistogram(int **dev_outputPixelMapping, const int *dev_cumulativeHistogram, int histogramRange) {
    hipError_t cudaStatus;

    cudaStatus = cudaCalloc(dev_outputPixelMapping, histogramRange * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_outputPixelMapping cudaCalloc failed!\n");
        return cudaStatus;
    }

    equalizeHistogram_kernel<<<1, histogramRange, histogramRange * sizeof(int)>>>(dev_outputPixelMapping[0], dev_cumulativeHistogram, histogramRange);

    // hipDeviceSynchronize waits for the kernel to finish
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching equalizeHistogram_kernel!\n", cudaStatus);
    }

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Failed to launch equalizeHistogram_kernel: %s\n", hipGetErrorString(cudaStatus));
    }

    return cudaStatus;
}

__global__ static void applyHistogramEqualizationOnImage_kernel(uchar *pixels, const int *pixelMapping, int size) {
    int start = (blockDim.x * blockIdx.x + threadIdx.x) * BATCH_DIM;
    int stop = start + BATCH_DIM;

    for (int i = start; (i < stop) && (i < size); ++i) {
        pixels[i] = pixelMapping[pixels[i]];
    }
}

static hipError_t applyHistogramEqualizationOnImage(uchar *outputPixels, uchar *dev_inputPixels, int *dev_pixelMapping, int rows, int cols) {
    hipError_t cudaStatus;
    hipStream_t streams[STREAM_COUNT];
    int blocks = ceil((double) rows / STREAM_COUNT);
    int threads = ceil((double) cols / BATCH_DIM);
    int size = 0;

    for (int i = 0; (i < STREAM_COUNT) && ((i * blocks) < rows); ++i) {
        hipStreamCreate(&streams[i]);

        if (i * blocks < rows) {
            size = (((i + 1) * blocks * cols) < (rows * cols) ? (blocks * cols) : ((rows - i * blocks) * cols));

            applyHistogramEqualizationOnImage_kernel<<<blocks, threads, 0, streams[i]>>>(&dev_inputPixels[i * blocks * cols], dev_pixelMapping, size);
        }
    }

    // hipDeviceSynchronize waits for the kernel to finish
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching applyHistogramEqualizationOnImage_kernel!\n", cudaStatus);
    }

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Failed to launch applyHistogramEqualizationOnImage_kernel: %s\n", hipGetErrorString(cudaStatus));
    }

    cudaStatus = hipMemcpy(outputPixels,
                            dev_inputPixels,
                            rows * cols * sizeof(uchar),
                            hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "outputPixels hipMemcpyAsync failed!\n");
        return cudaStatus;
    }

    for (int i = 0; (i < STREAM_COUNT) && ((i * blocks) < rows); ++i) {
        hipStreamDestroy(streams[i]);
    }

    return cudaStatus;
}

hipError_t cudaEnhanceContrast(uchar *outputPixels,
                                uchar **dev_inputPixels,
                                uchar *inputPixels,
                                int rows,
                                int cols,
                                int **dev_outputHistogram,
                                int **dev_outputCumulativeHistogram,
                                int **dev_outputPixelMapping,
                                int histogramRange) {
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (hipSuccess != cudaStatus) {
        fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?\n");
        return cudaStatus;
    }

    //Compute histogram
    cudaStatus = computeHistogram(dev_outputHistogram, histogramRange, dev_inputPixels, inputPixels, rows, cols);
    if (hipSuccess != cudaStatus) {
        fprintf(stderr, "Failed to computeHistogram\n");
        return cudaStatus;
    }

    //Compute cumulativeHistogram
    cudaStatus = computeCumulativeSum(dev_outputCumulativeHistogram, dev_outputHistogram[0], histogramRange);
    if (hipSuccess != cudaStatus) {
        fprintf(stderr, "Failed to computeCumulativeSum\n");
        return cudaStatus;
    }

    //Equalize histogram
    cudaStatus = equalizeHistogram(dev_outputPixelMapping, dev_outputCumulativeHistogram[0], histogramRange);
    if (hipSuccess != cudaStatus) {
        fprintf(stderr, "Failed to equalizeHistogram\n");
        return cudaStatus;
    }

    //Enhance contrast
    cudaStatus = applyHistogramEqualizationOnImage(outputPixels, dev_inputPixels[0], dev_outputPixelMapping[0], rows, cols);
    if (hipSuccess != cudaStatus) {
        fprintf(stderr, "Failed to applyHistogramEqualizationOnImage\n");
        return cudaStatus;
    }

    return cudaStatus;
}


hipError_t enhanceContrast(cv::Mat &outputImage, const cv::Mat &inputImage, int histogramRange) {
    uchar *dev_inputPixels;
    int *dev_histogram = nullptr;
    int *dev_cumulativeHistogram = nullptr;
    int *dev_pixelMapping = nullptr;
    hipError_t cudaStatus;

    //Enhance contrast
    cudaStatus = cudaEnhanceContrast(outputImage.data,
                                     &dev_inputPixels,
                                     inputImage.data,
                                     inputImage.rows,
                                     inputImage.cols,
                                     &dev_histogram,
                                     &dev_cumulativeHistogram,
                                     &dev_pixelMapping,
                                     histogramRange);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Failed to enhance contrast. Status = %s\n", hipGetErrorString(cudaStatus));
    }

    //free resources
    hipFree(dev_inputPixels);
    hipFree(dev_histogram);
    hipFree(dev_cumulativeHistogram);
    hipFree(dev_pixelMapping);

    return cudaStatus;
}

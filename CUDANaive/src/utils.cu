#include "utils.cuh"

#include <opencv2/imgproc.hpp>
#include <opencv2/highgui.hpp>
#include <iostream>

hipError_t runKernelWithCuda(void(*kernel)(void *, const void *, const void *),
                              void *out,
                              const void *in1,
                              const void *in2,
                              int outSizeInBytes,
                              int inSize1InBytes,
                              int inSize2InBytes,
                              int rows,
                              int cols) {
    void *dev_in1 = nullptr;
    void *dev_in2 = nullptr;
    void *dev_out = nullptr;
    hipError_t cudaStatus;

    auto start = std::chrono::high_resolution_clock::now();

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = cudaCalloc((void **) &dev_in1, inSize1InBytes);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = cudaCalloc((void **) &dev_in2, inSize2InBytes);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = cudaCalloc((void **) &dev_out, outSizeInBytes);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_in1, in1, inSize1InBytes, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_in2, in2, inSize2InBytes, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

//    auto finish = std::chrono::high_resolution_clock::now();
//    std::cout << "Transfer Host->Device GPU : "
//              << std::chrono::duration_cast<std::chrono::microseconds>(finish - start).count() << "microsec\n";

    // Launch a kernel on the GPU with one thread for each element.
    kernel <<<rows, cols >>>(dev_out, dev_in1, dev_in2);

    start = std::chrono::high_resolution_clock::now();

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(out, dev_out, outSizeInBytes, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

//    finish = std::chrono::high_resolution_clock::now();
//    std::cout << "Transfer Device->Host GPU : "
//              << std::chrono::duration_cast<std::chrono::microseconds>(finish - start).count() << "microsec\n";

    Error:
    hipFree(dev_in1);
    hipFree(dev_in2);
    hipFree(dev_out);

    return cudaStatus;
}